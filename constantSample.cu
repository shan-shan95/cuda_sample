#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__constant__ int cut_con[81] = {
  5,2,1,1,1,1,1,2,5,
  2,1,0,0,0,0,0,1,2,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  2,1,0,0,0,0,0,1,2,
  5,2,1,1,1,1,1,2,5
};

__global__ void culCellConstant(int nx, int ny, int nz) {

  int cut_num;

  if (threadIdx.x < nx && threadIdx.y < ny && threadIdx.z < nz) {
    for (int x = 0; x < 81; x++) {
      cut_num = cut_con[x];
    }
  }
}

int main(int argc, char **argv) {
  printf("%s Starting...\n", argv[0]);

  //行列のデータサイズを指定
  int nx = 1 << 10;
  int ny = 1 << 10;
  int nz = 1 << 10;

  int nxyz = nx * ny * nz;
  printf("Matrix size: nx %d ny %d nz %d\n", nx, ny, nz);

  //ホスト側でカーネルを呼び出す
  int dimx = 32;
  int dimy = 32;
  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

  // //コンスタントメモリ使用
  // culCellConstant<<< grid, block >>>(nx, ny, nz);
  // cudaDeviceSynchronize();

  //シェアドメモリ使用
  culCellConstant<<< grid, block >>>(nx, ny, nz);
  hipDeviceSynchronize();

  //カーネルエラーをチェック
  hipGetLastError();

  //デバイスのグローバルメモリを解放
  hipFree(cut_con);

  //デバイスをリセット
  hipDeviceReset();

  return(0);
}
