#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__constant__ int cut_con[81] = {
  5,2,1,1,1,1,1,2,5,
  2,1,0,0,0,0,0,1,2,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  2,1,0,0,0,0,0,1,2,
  5,2,1,1,1,1,1,2,5
};

__global__ void culCellConstant(int nx, int ny, int nz) {

  int cut_num;

  if (threadIdx.x < nx && threadIdx.y < ny && threadIdx.z < nz) {
    for (int x = 0; x < 81; x++) {
      cut_num = cut_con[x];
    }
  }
}

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

int main(int argc, char **argv) {
  hipEvent_t start, stop;
  float elapsed_time_ms;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  printf("%s Starting...\n", argv[0]);

  //行列のデータサイズを指定
  int nx = 1 << 7;
  int ny = 1 << 7;
  int nz = 1 << 7;

  printf("Matrix size: nx %d ny %d nz %d\n", nx, ny, nz);

  //ホスト側でカーネルを呼び出す
  int dimx = 128;
  int dimy = 128;
  int dimz = 1;
  dim3 block(dimx, dimy, dimz);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y, (nz + block.z - 1) / block.z);
  printf("grid: %d, %d, %d, block: %d, %d, %d\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);


  //コンスタントメモリ使用
  hipEventRecord(start, 0);
  culCellConstant<<< grid, block >>>(nx, ny, nz);
  hipEventRecord(stop, 0);
  hipDeviceSynchronize();
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  printf("time: %8.2f ms \n", elapsed_time_ms);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //カーネルエラーをチェック
  hipGetLastError();

  //デバイスのグローバルメモリを解放
  hipFree(cut_con);

  //デバイスをリセット
  hipDeviceReset();

  return(0);
}
