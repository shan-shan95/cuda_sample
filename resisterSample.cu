#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__global__ void culCellResister(int nx, int ny, int nz) {
  int cut_num;

  //実行時間155usほど。つまりシェアドメモリそのままと変わらない
  int cut_res[81] = {
    5,2,1,1,1,1,1,2,5,
    2,1,0,0,0,0,0,1,2,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    2,1,0,0,0,0,0,1,2,
    5,2,1,1,1,1,1,2,5
  };

  if (threadIdx.x < nx && threadIdx.y < ny && threadIdx.z < nz) {
    for (int x = 0; x < 81; x++) {
      cut_num = cut_res[x];
    }
  }
}

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

int main(int argc, char **argv) {
  hipEvent_t start, stop;
  float elapsed_time_ms;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  printf("%s Starting...\n", argv[0]);

  //行列のデータサイズを指定
  int nx = 1 << 10;
  int ny = 1 << 10;
  int nz = 1 << 10;

  printf("Matrix size: nx %d ny %d nz %d\n", nx, ny, nz);

  //ホスト側でカーネルを呼び出す
  int dimx = 32;
  int dimy = 32;
  int dimz = 1;
  dim3 block(dimx, dimy, dimz);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y, (nz + block.z - 1) / block.z);
  printf("grid: %d, %d, %d, block: %d, %d, %d\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

  //レジスタ使用
  hipEventRecord(start, 0);
  culCellResister<<< grid, block >>>(nx, ny, nz);
  hipEventRecord(stop, 0);
  hipDeviceSynchronize();
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  printf("time: %8.2f ms \n", elapsed_time_ms);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //カーネルエラーをチェック
  hipGetLastError();

  //デバイスをリセット
  hipDeviceReset();

  return(0);
}
