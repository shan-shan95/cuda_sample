#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__global__ void culCellResister(int nx, int ny, int nz) {
  int cut_num;

  //実行時間155usほど。つまりシェアドメモリそのままと変わらない
  int cut_res[81] = {
    5,2,1,1,1,1,1,2,5,
    2,1,0,0,0,0,0,1,2,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    2,1,0,0,0,0,0,1,2,
    5,2,1,1,1,1,1,2,5
  };

  if (threadIdx.x < nx && threadIdx.y < ny && threadIdx.z < nz) {
    for (int x = 0; x < 81; x++) {
      cut_num = cut_res[x];
    }
  }
}

int main(int argc, char **argv) {
  printf("%s Starting...\n", argv[0]);

  //行列のデータサイズを指定
  int nx = 1 << 10;
  int ny = 1 << 10;
  int nz = 1 << 10;

  int nxyz = nx * ny * nz;
  printf("Matrix size: nx %d ny %d nz %d\n", nx, ny, nz);

  //ホスト側でカーネルを呼び出す
  int dimx = 32;
  int dimy = 32;
  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

  //レジスタ使用
  culCellResister<<< grid, block >>>(nx, ny, nz);
  hipDeviceSynchronize();

  //カーネルエラーをチェック
  hipGetLastError();
  
  //デバイスをリセット
  hipDeviceReset();

  return(0);
}
