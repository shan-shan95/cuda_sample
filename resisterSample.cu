#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__global__ void culCellResister(int nx, int ny, int nz) {

  //実行時間155usほど。つまりシェアドメモリそのままと変わらない
  //カーネル内で変数宣言すればレジスタに乗ると思っていたが、間違えていた
  //配列はキャッシュに乗らない。おそらくコンスタントメモリに乗っているので結果が同じになった？
}

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

int main(int argc, char **argv) {
  hipEvent_t start, stop;
  float elapsed_time_ms;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  printf("%s Starting...\n", argv[0]);

  //行列のデータサイズを指定
  int nx = 1 << 10;
  int ny = 1 << 10;
  int nz = 1 << 10;

  printf("Matrix size: nx %d ny %d nz %d\n", nx, ny, nz);

  //ホスト側でカーネルを呼び出す
  int dimx = 32;
  int dimy = 32;
  int dimz = 1;
  dim3 block(dimx, dimy, dimz);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y, (nz + block.z - 1) / block.z);
  printf("grid: %d, %d, %d, block: %d, %d, %d\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

  //レジスタ使用
  hipEventRecord(start, 0);
  culCellResister<<< grid, block >>>(nx, ny, nz);
  hipEventRecord(stop, 0);
  hipDeviceSynchronize();
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  printf("time: %8.2f ms \n", elapsed_time_ms);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //カーネルエラーをチェック
  hipGetLastError();

  //デバイスをリセット
  hipDeviceReset();

  return(0);
}
