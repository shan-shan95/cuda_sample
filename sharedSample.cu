#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(hipError_t call) {
  const hipError_t error = call;
  if (error != hipSuccess) {
    printf("Error: %s:%d, ", __FILE__, __LINE__);
    printf("code: %d, reason: %s\n", error, hipGetErrorString(error));
    exit(1);
  }
}

__constant__ int cut_con[81] = {
  5,2,1,1,1,1,1,2,5,
  2,1,0,0,0,0,0,1,2,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  2,1,0,0,0,0,0,1,2,
  5,2,1,1,1,1,1,2,5
};

__shared__ float *cut_sha;

__global__ void culCellConstant(int nx, int ny, int nz) {
  if (threadIdx.x < nx && threadIdx.y < ny && threadIdx.z < nz) {
    for (int x = 0; x < 81; x++) {
      int cut_num = cut_con[x];
    }
  }
}

int main(int argc, char **argv) {
  printf("%s Starting...\n", argv[0]);

  //行列のデータサイズを指定
  int nx = 1 << 9;
  int ny = 1 << 9;
  int nz = 1 << 9;

  int nxyz = nx * ny * nz;
  int nBytes = nxyz * sizeof(float);
  printf("Matrix size: nx %d ny %d nz %d\n", nx, ny, nz);

  //デバイスのコンスタントメモリを確保
  CHECK(hipMalloc((void **)&d_cut_con, 81 * sizeof(int)));

  //ホストからデバイスへデータを転送
  CHECK(hipMemcpy(d_cut_con, cut_con, 81 * sizeof(int), cudaMemcpyHostToDevise));

  //ホスト側でカーネルを呼び出す
  int dimx = 512;
  int dimy = 512;
  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

  iStart = cpuSecond();
  culCellConstant<<< grid, block >>>(nx, ny, nz);
  CHECK(cudaDeviseSynchronize());
  iElaps = cpuSecond() - iStart;
  printf("sumMatrixOnGPU2D <<<(%d, %d), (%d, %d)>>> elapsed %f sec\n",
          grid.x, grid.y, block.x, block.y, iElaps);
  //カーネルエラーをチェック
  CHECK(hipGetLastError());

  //デバイスのグローバルメモリを解放
  CHECK(hipFree(d_cut_con));

  //デバイスをリセット
  CHECK(hipDeviceReset());

  return(0);
}
