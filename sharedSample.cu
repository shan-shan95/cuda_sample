#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__shared__ float cut_sha[81];

__global__ void culCellShared(int nx, int ny, int nz) {
  int cut_num;

  //実行時間150msほど
  // cut_sha[0] = 5;
  // cut_sha[1] = 2;
  // cut_sha[2] = 1;
  // cut_sha[3] = 1;
  // cut_sha[4] = 1;
  // cut_sha[5] = 1;
  // cut_sha[6] = 1;
  // cut_sha[7] = 2;
  // cut_sha[8] = 5;
  // cut_sha[9] = 2;
  // cut_sha[10] = 1;
  // cut_sha[11] = 0;
  // cut_sha[12] = 0;
  // cut_sha[13] = 0;
  // cut_sha[14] = 0;
  // cut_sha[15] = 0;
  // cut_sha[16] = 1;
  // cut_sha[17] = 2;
  // cut_sha[18] = 1;
  // cut_sha[19] = 0;
  // cut_sha[20] = 0;
  // cut_sha[21] = 0;
  // cut_sha[22] = 0;
  // cut_sha[23] = 0;
  // cut_sha[24] = 0;
  // cut_sha[25] = 0;
  // cut_sha[26] = 1;
  // cut_sha[27] = 1;
  // cut_sha[28] = 0;
  // cut_sha[29] = 0;
  // cut_sha[30] = 0;
  // cut_sha[31] = 0;
  // cut_sha[32] = 0;
  // cut_sha[33] = 0;
  // cut_sha[34] = 0;
  // cut_sha[35] = 1;
  // cut_sha[36] = 1;
  // cut_sha[37] = 0;
  // cut_sha[38] = 0;
  // cut_sha[39] = 0;
  // cut_sha[40] = 0;
  // cut_sha[41] = 0;
  // cut_sha[42] = 0;
  // cut_sha[43] = 0;
  // cut_sha[44] = 1;
  // cut_sha[45] = 1;
  // cut_sha[46] = 0;
  // cut_sha[47] = 0;
  // cut_sha[48] = 0;
  // cut_sha[49] = 0;
  // cut_sha[50] = 0;
  // cut_sha[51] = 0;
  // cut_sha[52] = 0;
  // cut_sha[53] = 1;
  // cut_sha[54] = 1;
  // cut_sha[55] = 0;
  // cut_sha[56] = 0;
  // cut_sha[57] = 0;
  // cut_sha[58] = 0;
  // cut_sha[59] = 0;
  // cut_sha[60] = 0;
  // cut_sha[61] = 0;
  // cut_sha[62] = 1;
  // cut_sha[63] = 2;
  // cut_sha[64] = 1;
  // cut_sha[65] = 0;
  // cut_sha[66] = 0;
  // cut_sha[67] = 0;
  // cut_sha[68] = 0;
  // cut_sha[69] = 0;
  // cut_sha[70] = 1;
  // cut_sha[71] = 2;
  // cut_sha[72] = 5;
  // cut_sha[73] = 2;
  // cut_sha[74] = 1;
  // cut_sha[75] = 1;
  // cut_sha[76] = 1;
  // cut_sha[77] = 1;
  // cut_sha[78] = 1;
  // cut_sha[79] = 2;
  // cut_sha[80] = 5;

  int threadId = (threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x) % 32;

  if (threadId == 0 || threadId == 24) {
    cut_sha[3 * threadId] = 5;
    cut_sha[3 * threadId + 1] = 2;
    cut_sha[3 * threadId + 2] = 1;
  } else if (threadId == 2 || threadId == 26) {
    cut_sha[3 * threadId] = 1;
    cut_sha[3 * threadId + 1] = 2;
    cut_sha[3 * threadId + 2] = 5;
  } else if (threadId == 1 || threadId == 25) {
    cut_sha[3 * threadId] = 1;
    cut_sha[3 * threadId + 1] = 1;
    cut_sha[3 * threadId + 2] = 1;
  } else if (threadId == 3 || threadId == 21) {
    cut_sha[3 * threadId] = 2;
    cut_sha[3 * threadId + 1] = 1;
    cut_sha[3 * threadId + 2] = 0;
  } else if (threadId == 4 || threadId == 7 || threadId == 10 || threadId == 13 || threadId == 16 || threadId == 19 || threadId == 22) {
    cut_sha[3 * threadId] = 0;
    cut_sha[3 * threadId + 1] = 0;
    cut_sha[3 * threadId + 2] = 0;
  } else if (threadId == 6 || threadId == 9 || threadId == 12 || threadId == 15 || threadId == 18) {
    cut_sha[3 * threadId] = 1;
    cut_sha[3 * threadId + 1] = 0;
    cut_sha[3 * threadId + 2] = 0;
  } else if (threadId == 5 || threadId == 25) {
    cut_sha[3 * threadId] = 0;
    cut_sha[3 * threadId + 1] = 1;
    cut_sha[3 * threadId + 2] = 2;
  } else if (threadId == 8 || threadId == 11 || threadId == 14 || threadId == 17 || threadId == 20) {
    cut_sha[3 * threadId] = 0;
    cut_sha[3 * threadId + 1] = 0;
    cut_sha[3 * threadId + 2] = 1;
  }

  if (threadIdx.x < nx && threadIdx.y < ny && threadIdx.z < nz) {
    for (int x = 0; x < 81; x++) {
      cut_num = cut_sha[x];
    }
  }
}

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

int main(int argc, char **argv) {
  hipEvent_t start, stop;
  float elapsed_time_ms;

  //タイマー開始
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  printf("%s Starting...\n", argv[0]);

  //行列のデータサイズを指定
  int nx = 1 << 10;
  int ny = 1 << 10;
  int nz = 1 << 10;

  printf("Matrix size: nx %d ny %d nz %d\n", nx, ny, nz);

  //ホスト側でカーネルを呼び出す
  int dimx = 32;
  int dimy = 32;
  int dimz = 1;
  dim3 block(dimx, dimy, dimz);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y, (nz + block.z - 1) / block.z);
  printf("grid: %d, %d, %d, block: %d, %d, %d\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

  //シェアドメモリ使用
  for(int i = 0 ; i < 1000 ; i++) {
    culCellShared<<< grid, block >>>(nx, ny, nz);
  }
  hipDeviceSynchronize();

  //カーネルエラーをチェック
  hipGetLastError();

  //タイマーをストップ
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  printf("time: %8.2f ms \n", elapsed_time_ms);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //デバイスをリセット
  hipDeviceReset();

  return(0);
}
