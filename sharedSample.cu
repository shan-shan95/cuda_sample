#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__constant__ int cut_con[81] = {
  5,2,1,1,1,1,1,2,5,
  2,1,0,0,0,0,0,1,2,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  1,0,0,0,0,0,0,0,1,
  2,1,0,0,0,0,0,1,2,
  5,2,1,1,1,1,1,2,5
};

__global__ void culCellConstant(int nx, int ny, int nz) {
  if (threadIdx.x < nx && threadIdx.y < ny && threadIdx.z < nz) {
    for (int x = 0; x < 81; x++) {
      int cut_num = cut_con[x];
    }
  }
}

int main(int argc, char **argv) {
  printf("%s Starting...\n", argv[0]);

  //行列のデータサイズを指定
  int nx = 1 << 9;
  int ny = 1 << 9;
  int nz = 1 << 9;

  int nxyz = nx * ny * nz;
  int nBytes = nxyz * sizeof(float);
  printf("Matrix size: nx %d ny %d nz %d\n", nx, ny, nz);

  //ホスト側でカーネルを呼び出す
  int dimx = 512;
  int dimy = 512;
  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

  iStart = cpuSecond();
  culCellConstant<<< grid, block >>>(nx, ny, nz);
  cudaDeviseSynchronize();
  iElaps = cpuSecond() - iStart;
  printf("sumMatrixOnGPU2D <<<(%d, %d), (%d, %d)>>> elapsed %f sec\n",
  grid.x, grid.y, block.x, block.y, iElaps);
  //カーネルエラーをチェック
  hipGetLastError();

  //デバイスのグローバルメモリを解放
  hipFree(cut_con);

  //デバイスをリセット
  hipDeviceReset();

  return(0);
}
