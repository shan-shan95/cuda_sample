#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call) {
  const hipError_t error = call;
  if (error != hipSuccess)
  {
    printf("Error: %s:%d, ", __FILE__, __LINE__);
    printf("code: %d, reason: %s\n", error, hipGetErrorString(error));
    exit(1);
  }
}

__constant__ float cut[262144] = {1};

__shared__ float cut2[262144];

void initialData(float *ip, int size) {
  //乱数シードを生成
  time_t t;
  srand((unsigned) time(&t));

  for (int i = 0; i < size; i++) {
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }

  return;
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny) {
  float *ia = A;
  float *ib = B;
  float *ic = C;

  for (int iy = 0; iy < ny; iy++) {
    for (int ix = 0; ix < nx; ix++) {
      ic[ix] = ia[ix] + ib[ix];
    }
    ia += nx;
    ib += nx;
    ic += nx;
  }

  return;
}

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny) {
  unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int idx = iy * nx + ix;

  if (ix < nx && iy < ny)
    MatC[idx] = cut[idx] + cut[idx];
}

__global__ void sumMatrixOnGPU2Dshared(float *MatA, float *MatB, float *MatC, int nx, int ny) {
  unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int idx = iy * nx + ix;

  if (ix < nx && iy < ny)
    MatC[idx] = cut[idx] + cut2[idx];
}

int main(int argc, char **argv) {
  printf("%s Starting...\n", argv[0]);

  //行列のデータサイズを指定
  int nx = 1 << 9;
  int ny = 1 << 9;

  int nxy = nx * ny;
  int nBytes = nxy * sizeof(float);
  printf("Matrix size: nx %d ny %d\n", nx, ny);

  //ホストメモリを確保
  float *h_A, *h_B, *hostRef, *gpuRef;
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  hostRef = (float *)malloc(nBytes);
  gpuRef = (float *)malloc(nBytes);

  //ホスト側でデータを初期化
  double iStart = cpuSecond();
  initialData (h_A, nxy);
  initialData (h_B, nxy);
  double iElaps = cpuSecond() - iStart;

  memset(hostRef, 0, nBytes);
  memset(gpuRef, 0, nBytes);

  //結果をチェックする為にホスト側で行列を加算
  iStart = cpuSecond();
  sumMatrixOnHost (h_A, h_B, hostRef, nx, ny);
  iElaps = cpuSecond() - iStart;

  //デバイスのグローバルメモリを確保
  float *d_MatA, *d_MatB, *d_MatC;
  CHECK(hipMalloc((void **)&d_MatA, nBytes));
  CHECK(hipMalloc((void **)&d_MatB, nBytes));
  CHECK(hipMalloc((void **)&d_MatC, nBytes));

  //ホストからデバイスへデータを転送
  CHECK(hipMemcpy(d_MatA, h_A, nBytes, cudaMemcpyHostToDevise));
  CHECK(hipMemcpy(d_MatB, h_B, nBytes, cudaMemcpyHostToDevise));

  //ホスト側でカーネルを呼び出す
  int dimx = 32;
  int dimy = 32;
  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

  iStart = cpuSecond();
  sumMatrixOnGPU2D<<< grid, block >>>(d_MatA, d_MatB, d_MatC, nx, ny);
  CHECK(cudaDeviseSynchronize());
  iElaps = cpuSecond() - iStart;
  printf("sumMatrixOnGPU2D <<<(%d, %d), (%d, %d)>>> elapsed %f sec\n",
          grid.x, grid.y, block.x, block.y, iElaps);
  //カーネルエラーをチェック
  CHECK(hipGetLastError());

  //カーネルの結果をホスト側にコピー
  CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, cudaMemcpyDeviseToHost));

  //デバイスの結果をチェック
  checkResult(hostRef, gpuRef, nxy);

  //デバイスのグローバルメモリを解放
  CHECK(hipFree(d_MatA));
  CHECK(hipFree(d_MatB));
  CHECK(hipFree(d_MatC));

  //ホストのメモリを解放
  free(h_A);
  free(h_B);
  free(hostRef);
  free(gpuRef);

  //デバイスをリセット
  CHECK(cudaDeviseReset());

  return(0);
}
