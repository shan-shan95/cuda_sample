#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__shared__ float cut_sha[81];

__global__ void culCellShared(int nx, int ny, int nz) {
  int cut_num;

  if (threadIdx.x < nx && threadIdx.y < ny && threadIdx.z < nz) {
    unsigned int threadId = (threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x) % 32;

    if (threadId < 27) {
      if (threadId == 0 || threadId == 2 || threadId == 24 || threadId == 26) {
        unsigned int t = (threadId << 30) >> 30;
        cut_sha[3 * threadId + t] = 5;
        cut_sha[3 * threadId + 1] = 2;
        cut_sha[3 * threadId + 2 - t] = 1;
      } else if (threadId == 1 || threadId == 25) {
        cut_sha[3 * threadId] = 1;
        cut_sha[3 * threadId + 1] = 1;
        cut_sha[3 * threadId + 2] = 1;
      } else if (threadId == 3 || threadId == 5 || threadId == 21 || threadId == 23) {
        unsigned int t = threadId % 3;
        cut_sha[3 * threadId + t] = 2;
        cut_sha[3 * threadId + 1] = 1;
        cut_sha[3 * threadId + 2 - t] = 0;
      } else if (threadId % 3 == 1 ) {
        cut_sha[3 * threadId] = 0;
        cut_sha[3 * threadId + 1] = 0;
        cut_sha[3 * threadId + 2] = 0;
      } else {
        unsigned int t = threadId % 3;
        cut_sha[3 * threadId + t] = 1;
        cut_sha[3 * threadId + 1] = 0;
        cut_sha[3 * threadId + 2 - t] = 0;
      }
      for (int x = 0; x < 81; x++) {
        cut_num = cut_sha[x];
      }
    }
  }
}

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

int main(int argc, char **argv) {
  hipEvent_t start, stop;
  float elapsed_time_ms;

  //タイマー開始
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  printf("%s Starting...\n", argv[0]);

  //行列のデータサイズを指定
  int nx = 1 << 10;
  int ny = 1 << 10;
  int nz = 1 << 10;

  printf("Matrix size: nx %d ny %d nz %d\n", nx, ny, nz);

  //ホスト側でカーネルを呼び出す
  int dimx = 32;
  int dimy = 32;
  int dimz = 1;
  dim3 block(dimx, dimy, dimz);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y, (nz + block.z - 1) / block.z);
  printf("grid: %d, %d, %d, block: %d, %d, %d\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

  //シェアドメモリ使用
  for(int i = 0 ; i < 1000 ; i++) {
    culCellShared<<< grid, block >>>(nx, ny, nz);
    hipDeviceSynchronize();
  }

  //タイマーをストップ
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  printf("time: %8.2f ms \n", elapsed_time_ms);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //カーネルエラーをチェック
  hipGetLastError();

  //デバイスをリセット
  hipDeviceReset();

  return(0);
}
