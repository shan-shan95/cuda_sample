#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__shared__ float cut_sha[81];

__global__ void culCellConstant(int nx, int ny, int nz) {
  __constant__ int cut_con[81] = {
    5,2,1,1,1,1,1,2,5,
    2,1,0,0,0,0,0,1,2,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    1,0,0,0,0,0,0,0,1,
    2,1,0,0,0,0,0,1,2,
    5,2,1,1,1,1,1,2,5
  };

  int cut_num;

  if (threadIdx.x < nx && threadIdx.y < ny && threadIdx.z < nz) {
    for (int x = 0; x < 81; x++) {
      cut_num = cut_con[x];
    }
  }
}

__global__ void culCellShared(int nx, int ny, int nz) {
  int cut_num;

  //実行時間155usほど
  cut_sha[0] = 5;
  cut_sha[1] = 2;
  cut_sha[2] = 1;
  cut_sha[3] = 1;
  cut_sha[4] = 1;
  cut_sha[5] = 1;
  cut_sha[6] = 1;
  cut_sha[7] = 2;
  cut_sha[8] = 5;
  cut_sha[9] = 2;
  cut_sha[10] = 1;
  cut_sha[11] = 0;
  cut_sha[12] = 0;
  cut_sha[13] = 0;
  cut_sha[14] = 0;
  cut_sha[15] = 0;
  cut_sha[16] = 1;
  cut_sha[17] = 2;
  cut_sha[18] = 1;
  cut_sha[19] = 0;
  cut_sha[20] = 0;
  cut_sha[21] = 0;
  cut_sha[22] = 0;
  cut_sha[23] = 0;
  cut_sha[24] = 0;
  cut_sha[25] = 0;
  cut_sha[26] = 1;
  cut_sha[27] = 1;
  cut_sha[28] = 0;
  cut_sha[29] = 0;
  cut_sha[30] = 0;
  cut_sha[31] = 0;
  cut_sha[32] = 0;
  cut_sha[33] = 0;
  cut_sha[34] = 0;
  cut_sha[35] = 1;
  cut_sha[36] = 1;
  cut_sha[37] = 0;
  cut_sha[38] = 0;
  cut_sha[39] = 0;
  cut_sha[40] = 0;
  cut_sha[41] = 0;
  cut_sha[42] = 0;
  cut_sha[43] = 0;
  cut_sha[44] = 1;
  cut_sha[45] = 1;
  cut_sha[46] = 0;
  cut_sha[47] = 0;
  cut_sha[48] = 0;
  cut_sha[49] = 0;
  cut_sha[50] = 0;
  cut_sha[51] = 0;
  cut_sha[52] = 0;
  cut_sha[53] = 1;
  cut_sha[54] = 1;
  cut_sha[55] = 0;
  cut_sha[56] = 0;
  cut_sha[57] = 0;
  cut_sha[58] = 0;
  cut_sha[59] = 0;
  cut_sha[60] = 0;
  cut_sha[61] = 0;
  cut_sha[62] = 1;
  cut_sha[63] = 2;
  cut_sha[64] = 1;
  cut_sha[65] = 0;
  cut_sha[66] = 0;
  cut_sha[67] = 0;
  cut_sha[68] = 0;
  cut_sha[69] = 0;
  cut_sha[70] = 1;
  cut_sha[71] = 2;
  cut_sha[72] = 5;
  cut_sha[73] = 2;
  cut_sha[74] = 1;
  cut_sha[75] = 1;
  cut_sha[76] = 1;
  cut_sha[77] = 1;
  cut_sha[78] = 1;
  cut_sha[79] = 2;
  cut_sha[80] = 5;

  // 実行時間200usほど
  // int threadIdxInWarp = (threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x) % 32;
  // switch (threadIdxInWarp){
  //   case 0:
  //     cut_sha[0] = 5;
  //     cut_sha[8] = 5;
  //     break;
  //   case 1:
  //     cut_sha[72] = 5;
  //     cut_sha[80] = 5;
  //     break;
  //   case 2:
  //     cut_sha[1] = 2;
  //     cut_sha[9] = 2;
  //     break;
  //   case 3:
  //     cut_sha[7] = 2;
  //     cut_sha[17] = 2;
  //     break;
  //   case 4:
  //     cut_sha[63] = 2;
  //     cut_sha[73] = 2;
  //     break;
  //   case 5:
  //     cut_sha[71] = 2;
  //     cut_sha[79] = 2;
  //     break;
  //   case 6:
  //     cut_sha[2] = 1;
  //     cut_sha[10] = 1;
  //     cut_sha[18] = 1;
  //     break;
  //   case 7:
  //     cut_sha[6] = 1;
  //     cut_sha[16] = 1;
  //     cut_sha[26] = 1;
  //     break;
  //   case 8:
  //     cut_sha[54] = 1;
  //     cut_sha[64] = 1;
  //     cut_sha[74] = 1;
  //     break;
  //   case 9:
  //     cut_sha[62] = 1;
  //     cut_sha[70] = 1;
  //     cut_sha[78] = 1;
  //     break;
  //   case 10:
  //     cut_sha[3] = 1;
  //     cut_sha[4] = 1;
  //     cut_sha[5] = 1;
  //     break;
  //   case 11:
  //     cut_sha[27] = 1;
  //     cut_sha[36] = 1;
  //     cut_sha[45] = 1;
  //     break;
  //   case 12:
  //     cut_sha[35] = 1;
  //     cut_sha[44] = 1;
  //     cut_sha[53] = 1;
  //     break;
  //   case 13:
  //     cut_sha[75] = 1;
  //     cut_sha[76] = 1;
  //     cut_sha[77] = 1;
  //     break;
  //   default:
  //     cut_sha[11] = 0;
  //     cut_sha[12] = 0;
  //     cut_sha[13] = 0;
  //     cut_sha[14] = 0;
  //     cut_sha[15] = 0;
  //     cut_sha[19] = 0;
  //     cut_sha[20] = 0;
  //     cut_sha[21] = 0;
  //     cut_sha[22] = 0;
  //     cut_sha[23] = 0;
  //     cut_sha[24] = 0;
  //     cut_sha[25] = 0;
  //     cut_sha[28] = 0;
  //     cut_sha[29] = 0;
  //     cut_sha[30] = 0;
  //     cut_sha[31] = 0;
  //     cut_sha[32] = 0;
  //     cut_sha[33] = 0;
  //     cut_sha[34] = 0;
  //     cut_sha[37] = 0;
  //     cut_sha[38] = 0;
  //     cut_sha[39] = 0;
  //     cut_sha[40] = 0;
  //     cut_sha[41] = 0;
  //     cut_sha[42] = 0;
  //     cut_sha[43] = 0;
  //     cut_sha[46] = 0;
  //     cut_sha[47] = 0;
  //     cut_sha[48] = 0;
  //     cut_sha[49] = 0;
  //     cut_sha[50] = 0;
  //     cut_sha[51] = 0;
  //     cut_sha[52] = 0;
  //     cut_sha[55] = 0;
  //     cut_sha[56] = 0;
  //     cut_sha[57] = 0;
  //     cut_sha[58] = 0;
  //     cut_sha[59] = 0;
  //     cut_sha[60] = 0;
  //     cut_sha[61] = 0;
  //     cut_sha[65] = 0;
  //     cut_sha[66] = 0;
  //     cut_sha[67] = 0;
  //     cut_sha[68] = 0;
  //     cut_sha[69] = 0;
  //     break;
  // }

  if (threadIdx.x < nx && threadIdx.y < ny && threadIdx.z < nz) {
    for (int x = 0; x < 81; x++) {
      cut_num = cut_sha[x];
    }
  }
}

int main(int argc, char **argv) {
  printf("%s Starting...\n", argv[0]);

  //行列のデータサイズを指定
  int nx = 1 << 10;
  int ny = 1 << 10;
  int nz = 1 << 10;

  int nxyz = nx * ny * nz;
  printf("Matrix size: nx %d ny %d nz %d\n", nx, ny, nz);

  //ホスト側でカーネルを呼び出す
  int dimx = 32;
  int dimy = 32;
  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

  //コンスタントメモリ使用
  culCellConstant<<< grid, block >>>(nx, ny, nz);
  hipDeviceSynchronize();

  //シェアドメモリ使用
  culCellShared<<< grid, block >>>(nx, ny, nz);
  hipDeviceSynchronize();

  //カーネルエラーをチェック
  hipGetLastError();

  //デバイスのグローバルメモリを解放
  hipFree(cut_con);

  //デバイスをリセット
  hipDeviceReset();

  return(0);
}
