#include "hip/hip_runtime.h"
__global__ void mathKernel1(float *c) {
  int tid =blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;

  if (tid % 2 == 0) {
    ia = 100.0f;
  } else {
    ib = 200.0f;
  }
  c[tid] = ia + ib;
}

__global__ void mathKernel2(float *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;

  if ((tid / wrapSize) % 2 == 0) {
    ia = 100.0f;
  } else {
    ib = 200.0f;
  }
  c[tid] = ia + ib;
}

int main(int argc, char **argv) {
  //デバイスのセットアップ
  int dev = 0;
  hipDeviceProp_t deviceProp;
  // 途中
  CHECK(hipGetDevice)
}
