int main(int argc, char **argv) {
  printf("%s Starting...\n", argv[0]);

  //デバイスのセットアップ
  int dev = 0;
  cudaDeviseProp deviseProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using Devise %d: %s\n", dev, deviseProp.name);
  CHECK(hipSetDevice(dev));

  //行列のデータサイズを指定
  int nx = 1 << 14;
  int ny = 1 << 14;

  int nxy = nx * ny;
  int nBytes = nxy * sizeof(float);
  printf("Matrix size: nx %d ny %d\n", nx, ny);

  //ホストメモリを確保
  
