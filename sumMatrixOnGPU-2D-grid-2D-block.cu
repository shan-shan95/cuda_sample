#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)
{
  const hipError_t error = call;
  if (error != hipSuccess)
  {
    printf("Error: %s:%d, ", __FILE__, __LINE__);
    printf("code: %d, reason: %s\n", error, hipGetErrorString(error));
    exit(1);
  }
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny) {
  float *ia = A;
  float *ib = B;
  float *ic = C;

  for (int iy = 0; iy < ny; iy++) {
    for (int ix = 0; ix < nx; ix++) {
      ic[ix] = ia[ix] + ib[ix];
    }
    ia += nx;
    ib += nx;
    ic += nx;
  }

  return;
}

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny) {
  unsigned int ix = threadIdx.x + blockIdx.x * BlockDim.x;
  unsigned int iy = threadIdx.y + blockIdx.y * BlockDim.y;
  unsigned int idx = iy * nx + ix;

  if (ix < nx && iy < ny)
    MatC[idx] = MatA[idx] + MatB[idx];
}

int main(int argc, char **argv) {
  printf("%s Starting...\n", argv[0]);

  //デバイスのセットアップ
  int dev = 0;
  cudaDeviseProp deviseProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using Devise %d: %s\n", dev, deviseProp.name);
  CHECK(hipSetDevice(dev));

  //行列のデータサイズを指定
  int nx = 1 << 14;
  int ny = 1 << 14;

  int nxy = nx * ny;
  int nBytes = nxy * sizeof(float);
  printf("Matrix size: nx %d ny %d\n", nx, ny);

  //ホストメモリを確保
  float *h_A, *h_B, *hostRef, *gpuRef;
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  hostRef = (float *)malloc(nBytes);
  gpuRef = (float *)malloc(nBytes);

  //ホスト側でデータを初期化
  double iStart = cpuSecond();
  initialData (h_A, nxy);
  initialData (h_B, nxy);
  double iElaps = cpuSecond() - iStart;

  memset(hostRef, 0, nBytes);
  memset(gpuRef, 0, nBytes);

  //結果をチェックする為にホスト側で行列を加算
  iStart = cpuSecond();
  sumMatrixOnHost (h_A, h_B, hostRef, nx, ny);
  iElaps = cpuSecond() - iStart;

  //デバイスのグローバルメモリを確保
  float *d_MatA, *d_MatB, *d_MatC;
  CHECK(hipMalloc((void **)&d_MatA, nBytes));
  CHECK(hipMalloc((void **)&d_MatB, nBytes));
  CHECK(hipMalloc((void **)&d_MatC, nBytes));

  //ホストからデバイスへデータを転送
  CHECK(hipMemcpy(d_MatA, h_A, nBytes, cudaMemcpyHostToDevise));
  CHECK(hipMemcpy(d_MatB, h_B, nBytes, cudaMemcpyHostToDevise));

  //ホスト側でカーネルを呼び出す
  int dimx = 32;
  int dimy = 32;
  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

  iStart = cpuSecond();
  sumMatrixOnGPU2D<<< grid, block >>>(d_MatA, d_MatB, d_MatC, nx, ny);
  CHECK(cudaDeviseSynchronize());
  iElaps = cpuSecond() - iStart;
  printf("sumMatrixOnGPU2D <<<(%d, %d), (%d, %d)>>> elapsed %f sec\n",
          grid.x, grid.y, block.x, block.y, iElaps);
  //カーネルエラーをチェック
  CHECK(hipGetLastError());

  //カーネルの結果をホスト側にコピー
  CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, cudaMemcpyDeviseToHost));

  //デバイスの結果をチェック
  checkResult(hostRef, gpuRef, nxy);

  //デバイスのグローバルメモリを解放
  CHECK(hipFree(d_MatA));
  CHECK(hipFree(d_MatB));
  CHECK(hipFree(d_MatC));

  //ホストのメモリを解放
  free(h_A);
  free(h_B);
  free(hostRef);
  free(gpuRef);

  //デバイスをリセット
  CHECK(cudaDeviseReset());

  return(0);
}
